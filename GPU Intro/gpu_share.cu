
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdlib>
#include <iostream>


#define BLOCK_SIZE 1024

#ifndef RADIUS
	#define RADIUS 3
#endif

#ifndef ITERS
	#define ITERS 100
#endif

#ifndef USE_L2
	#define USE_L2 false 
#endif


__global__ void stencil_no_shared(int *in, int *out)
{
	int temp[BLOCK_SIZE + 2 * RADIUS];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x + RADIUS;
	// Read input elements into shared memory
	temp[lindex] = in[gindex+RADIUS];
	if (threadIdx.x < RADIUS) {
		temp[lindex - RADIUS] = in[gindex];
		temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE + RADIUS];
	}
	 __syncthreads();
	// Apply the stencil
	int result = 0;
	for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
	result += temp[lindex + offset];
	// Store the result
	out[gindex] = result;

}

__global__ void stencil_shared(int *in, int *out)
{
	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x + RADIUS;
	// Read input elements into shared memory
	temp[lindex] = in[gindex+RADIUS];
	if (threadIdx.x < RADIUS) {
		temp[lindex - RADIUS] = in[gindex];
		temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE + RADIUS];
	}
	 __syncthreads();
	// Apply the stencil
	int result = 0;
	for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
	result += temp[lindex + offset];
	// Store the result
	out[gindex] = result;

}


int main(int argc, const char *argv[])
{
	using namespace std::chrono;

	int iters;
	if (argc < 2) {
		iters = ITERS;
	} else {
		iters = std::atoi(argv[1]);
	}

	int *in  = static_cast<int *>(aligned_alloc(64, (BLOCK_SIZE+2*RADIUS) * sizeof(int)));
	int *out  = static_cast<int *>(aligned_alloc(64, (BLOCK_SIZE) * sizeof(int)));

	for(int i=0; i<BLOCK_SIZE+2*RADIUS; i++)
	{
		in[i]=i;
	}

	//copying mem to device
	int *d_in_ns;
	int *d_in_s;
	int *d_out_ns;
	int *d_out_s;
	hipMalloc(&d_in_ns,	(BLOCK_SIZE+2*RADIUS) * sizeof(int));
	hipMalloc(&d_in_s, 	(BLOCK_SIZE+2*RADIUS) * sizeof(int));
	hipMalloc(&d_out_ns, 	BLOCK_SIZE * sizeof(int));
	hipMalloc(&d_out_s, 	BLOCK_SIZE * sizeof(int));

	hipMemcpy(d_in_ns,	in,	(BLOCK_SIZE+2*RADIUS) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_in_s,	in,	(BLOCK_SIZE+2*RADIUS) * sizeof(int), hipMemcpyHostToDevice);
	//cudaMemcpy(d_out_ns,out,BLOCK_SIZE * sizeof(int), cudaMemcpyHostToDevice);
	//cudaMemcpy(d_out_s,	out,BLOCK_SIZE * sizeof(int), cudaMemcpyHostToDevice);


	//run without using shared cache
 	auto start = high_resolution_clock::now();
 	for(auto i=0; i<iters; i++)
 	{
 		#if (USE_L2==true)
 			stencil_no_shared<<<BLOCK_SIZE, 1>>>(d_in_s, d_out_s);
 		#else
 			stencil_no_shared<<<1, BLOCK_SIZE>>>(d_in_s, d_out_s);
 		#endif
 		hipDeviceSynchronize();
 	}
 	auto end = high_resolution_clock::now();
	auto diff = duration_cast<microseconds>(end - start);

	#if (USE_L2==true)
		std::cout << "CUDA stencil with L2 cache:\n"
	#else
		std::cout << "CUDA stencil with L1 cache:\n"
	#endif
			<< "\tNumber of Elements: " << BLOCK_SIZE << "\n"
			<< "\tRadius: " << RADIUS << "\n"
			<< "\tIterations = " << iters << "\n"
			<< "\tAverage time = " << diff.count() / iters
			<< " micros. Total time= " << diff.count() << " micros.\n";

	//run with share cache
	start = high_resolution_clock::now();
 	for(auto i=0; i<iters; i++)
 	{
 		stencil_shared<<<1, BLOCK_SIZE>>>(d_in_s, d_out_s);
 		hipDeviceSynchronize();
 	}
 	end = high_resolution_clock::now();
	diff = duration_cast<microseconds>(end - start);

	std::cout << "CUDA stencil with shared cache:\n"
			<< "\tNumber of Elements: " << BLOCK_SIZE << "\n"
			<< "\tRadius: " << RADIUS << "\n"
			<< "\tIterations = " << iters << "\n"
			<< "\tAverage time = " << diff.count() / iters
			<< " micros. Total time= " << diff.count() << " micros.\n\n";

	return 0;


}