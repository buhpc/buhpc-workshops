#include <stdio.h>


void deviceQuery ()
{
  hipDeviceProp_t prop;
  int nDevices=0, i;
  hipError_t ierr;

  ierr = hipGetDeviceCount(&nDevices);
  if (ierr != hipSuccess) { printf("Sync error: %s\n", hipGetErrorString(ierr)); }



  for( i = 0; i < nDevices; ++i )
  {
     ierr = hipGetDeviceProperties(&prop, i);
     printf("Device number: %d\n", i);
     printf("  Device name: %s\n", prop.name);
     printf("  Compute capability: %d.%d\n\n", prop.major, prop.minor);
     
     printf("  Clock Rate: %d kHz\n", prop.clockRate);
     printf("  Total SMs: %d \n", prop.multiProcessorCount);
     printf("  Shared Memory Per SM: %lu bytes\n", prop.sharedMemPerMultiprocessor);
     printf("  Registers Per SM: %d 32-bit\n", prop.regsPerMultiprocessor);
     printf("  Max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
     printf("  L2 Cache Size: %d bytes\n", prop.l2CacheSize);
     printf("  Total Global Memory: %lu bytes\n", prop.totalGlobalMem);
     printf("  Memory Clock Rate: %d kHz\n\n", prop.memoryClockRate);
     
     
     printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
     printf("  Max threads in X-dimension of block: %d\n", prop.maxThreadsDim[0]);
     printf("  Max threads in Y-dimension of block: %d\n", prop.maxThreadsDim[1]);
     printf("  Max threads in Z-dimension of block: %d\n\n", prop.maxThreadsDim[2]);

     printf("  Max blocks in X-dimension of grid: %d\n", prop.maxGridSize[0]);
     printf("  Max blocks in Y-dimension of grid: %d\n", prop.maxGridSize[1]);
     printf("  Max blocks in Z-dimension of grid: %d\n\n", prop.maxGridSize[2]);     
     
     printf("  Shared Memory Per Block: %lu bytes\n", prop.sharedMemPerBlock);
     printf("  Registers Per Block: %d 32-bit\n", prop.regsPerBlock);
     printf("  Warp size: %d\n\n", prop.warpSize);

  }
}

int main() {
    deviceQuery();
}