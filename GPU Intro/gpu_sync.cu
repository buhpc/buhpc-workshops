
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdlib>
#include <iostream>


#ifndef RADIUS
	#define RADIUS 5
#endif

#ifndef BLOCK_SIZE
	#define BLOCK_SIZE 10
#endif


__global__ void stencil_no_sync(int *in, int *out)
{
	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x + RADIUS;
	// Read input elements into shared memory
	temp[lindex] = in[gindex+RADIUS];
	if (threadIdx.x < RADIUS) {
		temp[lindex - RADIUS] = in[gindex];
		temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE + RADIUS];
	}
	////////////////////////////// missing sync thread ////////////////////////
	
	// Apply the stencil
	int result = 0;
	for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
		result += temp[lindex + offset];
	// Store the result
	out[gindex] = result;

}

__global__ void stencil_sync(int *in, int *out)
{
	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x + RADIUS;
	// Read input elements into shared memory
	temp[lindex] = in[gindex+RADIUS];
	if (threadIdx.x < RADIUS) {
		temp[lindex - RADIUS] = in[gindex];
		temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE + RADIUS];
	}
	////////////////////////////////// sync thread ////////////////////////////
	 __syncthreads();

	// Apply the stencil
	int result = 0;
	for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
		result += temp[lindex + offset];
	// Store the result
	out[gindex] = result;

}


int main(int argc, const char *argv[])
{

	int *in  = static_cast<int *>(aligned_alloc(64, (BLOCK_SIZE+2*RADIUS) * sizeof(int)));
	int *out_ns  = static_cast<int *>(aligned_alloc(64, (BLOCK_SIZE) * sizeof(int)));
	int *out_s  = static_cast<int *>(aligned_alloc(64, (BLOCK_SIZE) * sizeof(int)));


	std::cout << "value of in array\n";

	for(int i=0; i<BLOCK_SIZE+2*RADIUS; i++)
	{
		in[i]=i+1;
		std::cout << in[i] << "\t";

	}
	std::cout << "\nCorrect answer\n";
	for(int i=RADIUS; i<BLOCK_SIZE+RADIUS; i++)
	{
		int sum=0;
		for(int j=i-RADIUS; j<=i+RADIUS; j++)
			sum+=in[j];
		std::cout << sum << "\t";
	}
	std::cout << "\n\n";
	

	//copying mem to device
	int *d_in;
	int *d_out_ns;
	int *d_out_s;
	hipMalloc(&d_in,	(BLOCK_SIZE+2*RADIUS) * sizeof(int));
	hipMalloc(&d_out_ns, 	BLOCK_SIZE * sizeof(int));
	hipMalloc(&d_out_s, 	BLOCK_SIZE * sizeof(int));
	hipMemcpy(d_in,	in,	(BLOCK_SIZE+2*RADIUS) * sizeof(int), hipMemcpyHostToDevice);

	//run without sync
	stencil_no_sync<<<1, BLOCK_SIZE>>>(d_in, d_out_ns);
	hipDeviceSynchronize();

	//get data
	hipMemcpy(out_ns,	d_out_ns,	(BLOCK_SIZE) * sizeof(int), hipMemcpyDeviceToHost);

	hipError_t __err = hipGetLastError();
	std::cout << "Result for stencil without __syncthreads: "<<hipGetErrorString(__err)<<"\n";
	for (auto i=0; i<BLOCK_SIZE; i++)
	{
		std::cout << out_ns[i] << "\t";
	}
	std::cout << "\n";


	//run with sync
	stencil_sync<<<1, BLOCK_SIZE>>>(d_in, d_out_s);
	hipDeviceSynchronize();

	//get data
	hipMemcpy(out_s,	d_out_s,	(BLOCK_SIZE) * sizeof(int), hipMemcpyDeviceToHost);

	__err = hipGetLastError();
	std::cout << "\nResult for stencil with __syncthreads: "<<hipGetErrorString(__err)<<"\n";
	for (auto i=0; i<BLOCK_SIZE; i++)
	{
		std::cout << out_s[i] << "\t";
	}
	std::cout << "\n\n";


	return 0;


}